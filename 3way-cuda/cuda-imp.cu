#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#define MAX_STRING_SIZE 2200
#define BATCH_SIZE 1000

int total_read = 0;

// checks errors
void checkCUDAError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// finds maax ascii
__global__ void findMaxASCII(int *d_out, char *d_in, int lines, int max_string_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= lines) return;

    char *line = &d_in[idx * max_string_size];
    int local_max = -1;

    for (int i = 0; i < max_string_size; i++) {
        char c = line[i];
        if (c == '\0') break;
        if (c > local_max) {
            local_max = c;
        }
    }

    d_out[idx] = local_max;
}

// Reads a batch of lines from a file
int readFile(FILE* fd, char linesArray[][MAX_STRING_SIZE]) {
    char buffer[MAX_STRING_SIZE];
    int count = 0;

    while (count < BATCH_SIZE && fgets(buffer, MAX_STRING_SIZE, fd)) {
        buffer[strcspn(buffer, "\n")] = 0;
        snprintf(linesArray[count], MAX_STRING_SIZE, "%s", buffer);
        count++;
    }
    total_read += count;
    return count;
}

// Prints results
void printResults(int* results, int totalLines,int offsetNum, FILE* fout) {
    for (int i = 0; i < totalLines; i++) {
        fprintf( fout, "Line %d: %d\n", offsetNum + i + 1, results[i]);
    }
}

int main(int argc, char *argv[]) {

    int threads_per_block = atoi(argv[1]);
    int blocks_per_grid = atoi(argv[2]);

    FILE* fd = fopen("/homes/dan/625/wiki_dump.txt", "r");

    char (*h_lines)[MAX_STRING_SIZE] = (char(*)[MAX_STRING_SIZE])malloc(BATCH_SIZE * MAX_STRING_SIZE * sizeof(char));
    int *h_max = (int*)malloc(BATCH_SIZE * sizeof(int));

    char *d_lines;
    int *d_max;

    checkCUDAError(hipMalloc((void**)&d_lines, BATCH_SIZE * MAX_STRING_SIZE * sizeof(char)), "malloc");
    checkCUDAError(hipMalloc((void**)&d_max, BATCH_SIZE * sizeof(int)), "malloc");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    dim3 dimBlock(threads_per_block);
    dim3 dimGrid(blocks_per_grid);

    int totalLines = 0, readLines;

    FILE* fout = fopen("./CudaOut.txt", "w");

    if (fout == NULL) 
    {
        printf("./OpenMPOut.txt");
        perror("fopen Failed for : ");
        return EXIT_FAILURE;
    }

    while ((readLines = readFile(fd, h_lines)) > 0) {
        checkCUDAError(hipMemcpy(d_lines, h_lines, readLines * MAX_STRING_SIZE * sizeof(char), hipMemcpyHostToDevice), "memcpy");
        findMaxASCII<<<dimGrid, dimBlock>>>(d_max, d_lines, readLines, MAX_STRING_SIZE);
        checkCUDAError(hipGetLastError(), "kernal");
        checkCUDAError(hipDeviceSynchronize(), "synchronize");

        checkCUDAError(hipMemcpy(h_max, d_max, readLines * sizeof(int), hipMemcpyDeviceToHost), "memcpy");
        printResults(h_max, readLines,totalLines, fout);
        totalLines += readLines;
    }

    // records end times
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    fprintf(fout, "Total lines processed: %d\n", totalLines);
    fprintf(fout, "Program finished in %.6f seconds.\n", milliseconds / 1000.0);
    printf("%.6f", milliseconds / 1000.0);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_lines);
    hipFree(d_max);
    free(h_lines);
    free(h_max);
    fclose(fd);

    return 0;
}