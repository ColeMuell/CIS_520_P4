#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#define MAX_STRING_SIZE 2200
#define BATCH_SIZE 1000

int total_read = 0;

// Function to check CUDA errors
void checkCUDAError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Kernel function to find the max ASCII character per line
__global__ void findMaxASCII(int *d_out, char *d_in, int lines, int max_string_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= lines) return;

    char *line = &d_in[idx * max_string_size];
    int local_max = -1;

    for (int i = 0; i < max_string_size; i++) {
        char c = line[i];
        if (c == '\0') break;
        if (c > local_max) {
            local_max = c;
        }
    }

    d_out[idx] = local_max;
}

// Reads a batch of lines from a file
int readFile(FILE* fd, char linesArray[][MAX_STRING_SIZE]) {
    char buffer[MAX_STRING_SIZE];
    int count = 0;

    while (count < BATCH_SIZE && fgets(buffer, MAX_STRING_SIZE, fd)) {
        buffer[strcspn(buffer, "\n")] = 0;
        snprintf(linesArray[count], MAX_STRING_SIZE, "%s", buffer);
        count++;
    }
    total_read += count;
    return count;
}

// Prints results
void printResults(int* results, int totalLines) {
    for (int i = 0; i < totalLines; i++) {
        printf("Line %d: %d\n", i, results[i]);
    }
}

// Main function with dynamic thread/block configuration
int main(int argc, char *argv[]) {

    int threads_per_block = atoi(argv[1]);
    int blocks_per_grid = atoi(argv[2]);

    FILE* fd = fopen("/homes/dan/625/wiki_dump.txt", "r");

    char (*h_lines)[MAX_STRING_SIZE] = (char(*)[MAX_STRING_SIZE])malloc(BATCH_SIZE * MAX_STRING_SIZE * sizeof(char));
    int *h_max = (int*)malloc(BATCH_SIZE * sizeof(int));

    char *d_lines;
    int *d_max;

    checkCUDAError(hipMalloc((void**)&d_lines, BATCH_SIZE * MAX_STRING_SIZE * sizeof(char)), "malloc");
    checkCUDAError(hipMalloc((void**)&d_max, BATCH_SIZE * sizeof(int)), "malloc");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    dim3 dimBlock(threads_per_block);
    dim3 dimGrid(blocks_per_grid);

    int totalLines = 0, readLines;
    
    while ((readLines = readFile(fd, h_lines)) > 0) {
        checkCUDAError(hipMemcpy(d_lines, h_lines, readLines * MAX_STRING_SIZE * sizeof(char), hipMemcpyHostToDevice), "memcpy");
        findMaxASCII<<<dimGrid, dimBlock>>>(d_max, d_lines, readLines, MAX_STRING_SIZE);
        checkCUDAError(hipGetLastError(), "kernal");
        checkCUDAError(hipDeviceSynchronize(), "synchronize");

        checkCUDAError(hipMemcpy(h_max, d_max, readLines * sizeof(int), hipMemcpyDeviceToHost), "memcpy");
        //printResults(h_max, readLines);
        totalLines += readLines;
    }

    // records end times
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

  //  printf("Total lines processed: %d\n", totalLines);
    //printf("Program finished in %.3f seconds.\n", milliseconds / 1000.0);
    printf("%.f", milliseconds / 1000.0);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_lines);
    hipFree(d_max);
    free(h_lines);
    free(h_max);
    fclose(fd);

    return 0;
}