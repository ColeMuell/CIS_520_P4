#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#define MAX_STRING_SIZE 2200
#define BATCH_SIZE 1000
#define THREADS_PER_BLOCK 256
int total_read = 0;

// Prints results. The param offset says how far off of 0 the batched lines are
void print_results(int max_ascii[], int offset, int count) {
    for (int i = 0; i < count; i++) {
        printf("Line %d: %d\n", i + offset, max_ascii[i]);
    }
}

// Host function to find the max ascii value in a given line
char find_max_host(const char* line, int length) {
    int max_int = -1;
    
    int len = strnlen(line, length);
    for (int i = 0; i < len; i++) {
        if (line[i] > max_int) {
            max_int = line[i];
        }
    }
    return max_int;
}

// kernal for finding max value in a line
__global__ void find_max_kernel(char* d_lines, int* d_max, int count, int max_string_size) {
    int line_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (line_idx < count) {
        char* line = &d_lines[line_idx * max_string_size];
        int max_int = -1;
    
        int len = 0;
        while (len < max_string_size && line[len] != '\0') {
            len++;
        }
        
        // Find max ASCII value in this line
        for (int i = 0; i < len; i++) {
            if (line[i] > max_int) {
                max_int = line[i];
            }
        }
        
        d_max[line_idx] = max_int;
    }
}

// Reads a batch of lines from file
int read_file(FILE* fd, char linesArray[][MAX_STRING_SIZE]) {
    char buffer[MAX_STRING_SIZE];
    int count = 0;
    
    while (count < BATCH_SIZE && fgets(buffer, MAX_STRING_SIZE, fd)) {
        buffer[strcspn(buffer, "\n")] = 0;
        snprintf(linesArray[count], MAX_STRING_SIZE, "%s", buffer);
        count++;
    }
    total_read += count;
    return count;
}

// Check CUDA errors
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "%s failed: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char **argv) {

    
    FILE* fd = fopen("/homes/dan/625/wiki_dump.txt", "r");
    
    if (fd == NULL) {
        perror("fopen Failed: ");
        return EXIT_FAILURE;
    }
    
    // allocate all the host memory
    char (*h_lines)[MAX_STRING_SIZE] = (char(*)[MAX_STRING_SIZE])malloc(sizeof(char) * BATCH_SIZE * MAX_STRING_SIZE);
    int *h_max = (int*)malloc(sizeof(int) * BATCH_SIZE);
    
    //checking to see if the host memory fails
    if (h_lines == NULL || h_max == NULL) {
        fprintf(stderr, "Host memory allocation failed!\n");
        exit(EXIT_FAILURE);
    }
    
    // allocate all the device memory
    char *d_lines;
    int *d_max;
    
    checkCudaError(hipMalloc((void**)&d_lines, sizeof(char) * BATCH_SIZE * MAX_STRING_SIZE), 
                  "hipMalloc for d_lines");
    checkCudaError(hipMalloc((void**)&d_max, sizeof(int) * BATCH_SIZE), 
                  "hipMalloc for d_max");
    
    int total_lines = 0;
    int read_lines;
    
    // Record timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    //important note to self, threads go into blocks, blocks go into grids. Usually want multiples of 32 threads, as that matches nicely with the GPU stuff
    
    while ((read_lines = read_file(fd, h_lines)) > 0) {
        // cpoy the batch from host to device
        checkCudaError(hipMemcpy(d_lines, h_lines, sizeof(char) * read_lines * MAX_STRING_SIZE, 
                               hipMemcpyHostToDevice), "hipMemcpy h_lines to d_lines");
        
        // calculate number of blocks based on batch size
        int blocks = (read_lines + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        
        // Launch kernel, the execution config is how many blocks and how many threads
        find_max_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_lines, d_max, read_lines, MAX_STRING_SIZE);
        
        // Check for kernel errors
        checkCudaError(hipGetLastError(), "CUDA kernel launch");
        checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize");
        
        // reuslsts to host
        checkCudaError(hipMemcpy(h_max, d_max, sizeof(int) * read_lines, hipMemcpyDeviceToHost), 
                      "hipMemcpy d_max to h_max");
        
        // print reuslts
        print_results(h_max, total_lines, read_lines);
        total_lines += read_lines;
    }
    
    // Record end time and calculate elapsed time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("Total lines processed: %d\n", total_lines);
    printf("Program finished in %.3f seconds.\n", milliseconds / 1000.0);
    
    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_lines);
    hipFree(d_max);
    fclose(fd);
    free(h_lines);
    free(h_max);
    
    return 0;
}